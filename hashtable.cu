#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <string.h>
#include <algorithm>
#include <iostream>
#include <omp.h>
#include <sys/time.h>
#include <tclap/CmdLine.h>
#include <hip/hip_runtime_api.h>
#include "hashtable.h"

__global__ void bucket_search(range_t* range,value_t* results,bucket_t* bucketArray,int* bucketIdxArray
){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int bucketIdx = tid / BUFFERSCALE;
    int idx = tid % BUFFERSCALE;
    bucket_t* bucket = &(bucketArray[bucketIdx]);
    uint32_t min = range->min;
    uint32_t max = range->max;
    _key_t key = bucket->kvArray[idx].key;
    results[tid] = (key >= min && key <= max) ? bucket->kvArray[idx].value : 0;
}

bool compare(const key_value_t& a, const key_value_t& b) {
    return a.key < b.key;
}

void BucketInsertByCPU(
    _key_t key, 
    value_t value, 
    bucket_t* bucketArray,
    int bucketId,
    int numInBucket,
    int bucketUsed,
    _key_t* newNodeKey)
{
    bucket_t* bucket = &(bucketArray[bucketId]);
    bucket->kvArray[numInBucket].key = key;
    bucket->kvArray[numInBucket].value = value;
    if (numInBucket + 1 == BUFFERSCALE) {
        //split
        std::sort(bucket->kvArray, bucket->kvArray + BUFFERSCALE, &compare);
        *newNodeKey = bucket->kvArray[BUFFERSCALE / 2].key;
        bucket_t* newBucket = &(bucketArray[bucketUsed]);
        memcpy(newBucket->kvArray, bucket->kvArray, sizeof(BUFFERSCALE * sizeof(key_value_t) / 2));
    }
}

int slRandomLevel(void) {
    int level = 1;
    while ((random()&0xFFFF) < (SKIPLIST_P * 0xFFFF))
        level += 1;
    return (level<GPU_SKIPLIST_MAXLEVEL) ? level : GPU_SKIPLIST_MAXLEVEL;
}

skiplistNode *slCreateNode(int level, _key_t key, int bufferId, int numInBucket = 1) {
    skiplistNode *node = (skiplistNode*)malloc(sizeof(skiplistNode)+level*sizeof(skiplistNode*));
    node->key = key;
    node->bucketId = bufferId;
    node->numInBucket = numInBucket;
    return node;
}

skiplist* slCreate(void) {
    int j;
    skiplist *sl;
    sl = (skiplist*)malloc(sizeof(*sl));

    //hipMalloc
    checkCudaErrors(hipMalloc((void**) &sl->bucketBuffer, BUFFERNUM * sizeof(bucket_t)));

    //malloc
    sl->bucketBufferCPU = (bucket_t*)malloc(BUFFERNUM * sizeof(bucket_t));

    sl->bufferUsed = 1;
    sl->level = 1;
    sl->length = 1;
    sl->header = slCreateNode(GPU_SKIPLIST_MAXLEVEL, 0, 0, 1);
    //BucketInsert<<<1, 1>>>(0, 0, sl->bucketBuffer);
    BucketInsertByCPU(0, 0, sl->bucketBufferCPU, 0, 0, 0, NULL);
    for (j = 0; j < GPU_SKIPLIST_MAXLEVEL; j++) {
        sl->header->level[j].forward = NULL;
    }
    sl->header->backward = NULL;
    sl->tail = NULL;
    return sl;
}

/* Free the specified skiplist node. The referenced SDS string representation
 * of the element is freed too, unless node->ele is set to NULL before calling
 * this function. */
void slFreeNode(skiplistNode *node) {
    free(node);
}

/* Free a whole skiplist. */
void slFree(skiplist *sl) {
    skiplistNode *node = sl->header->level[0].forward, *next;
    free(sl->header);
    while(node) {
        next = node->level[0].forward;
        slFreeNode(node);
        node = next;
    }
    checkCudaErrors(hipFree(sl->bucketBuffer));
    free(sl->bucketBufferCPU);

    free(sl);
}

skiplistNode* slSearchForNode(skiplist *sl, _key_t key) {
    skiplistNode *node = sl->header;
    for (int i = sl-> level - 1; i >= 0; i--) {
        while (node->level[i].forward && node->level[i].forward->key < key)
            node = node->level[i].forward;
    }
    return node;
}

void slInsert(skiplist *sl, _key_t key, value_t value) {
    skiplistNode* targetNode = slSearchForNode(sl, key);
    _key_t h_result = 0;
    if (targetNode->numInBucket < BUFFERSCALE - 1) {
        BucketInsertByCPU(key, value, sl->bucketBufferCPU, targetNode->bucketId, targetNode->numInBucket, sl->bufferUsed, &h_result);
        targetNode->numInBucket += 1;
    } else {
        // need to split the node into two nodes
        if (sl->bufferUsed == BUFFERNUM){
            printf("No more buffer can be used!\n");
            exit(1);
        }
        BucketInsertByCPU(key, value, sl->bucketBufferCPU, targetNode->bucketId, targetNode->numInBucket, sl->bufferUsed, &h_result);
        targetNode->numInBucket = BUFFERSCALE / 2;
        
        //create the new node and insert into skiplist
        skiplistNode *update[GPU_SKIPLIST_MAXLEVEL], *x;
        int i, level;

        x = sl->header;
        for (i = sl->level-1; i >= 0; i--) {
            while (x->level[i].forward && x->level[i].forward->key < h_result)
                x = x->level[i].forward;
            update[i] = x;
        }
        /* we assume the element is not already inside, since we allow duplicated
         * scores, reinserting the same element should never happen since the
         * caller of slInsert() should test in the hash table if the element is
         * already inside or not. */
        level = slRandomLevel();
        if (level > sl->level) {
            for (i = sl->level; i < level; i++) {
                update[i] = sl->header;
            }
            sl->level = level;
        }
        x = slCreateNode(level, h_result, sl->bufferUsed, BUFFERSCALE / 2);
        sl->bufferUsed += 1;
        for (i = 0; i < level; i++) {
            x->level[i].forward = update[i]->level[i].forward;
            update[i]->level[i].forward = x;
        }

        x->backward = (update[0] == sl->header) ? NULL : update[0];
        if (x->level[0].forward)
            x->level[0].forward->backward = x;
        else
            sl->tail = x;
        sl->length++;
    }
}

void BufferMemCpy(skiplist* sl) {
    checkCudaErrors(hipMemcpy(sl->bucketBuffer, sl->bucketBufferCPU, BUFFERNUM * sizeof(bucket_t), hipMemcpyHostToDevice));
}

void getIndexArray(skiplist* sl, _key_t keyMin, _key_t keyMax, int* bufferIndex, int groupSize) {
    skiplistNode* startNode = slSearchForNode(sl, keyMin);
    skiplistNode* endNode = slSearchForNode(sl, keyMax);
    int idx = 0, idxInArray = 0;
    for (skiplistNode* node = startNode; node != endNode->level[0].forward; node = node->level[0].forward) {
        idxInArray += BUFFERSCALE;
        if (idxInArray >= groupSize)break;
        bufferIndex[idx++] = node->bucketId;
    }
}

int main(int argc, char* argv[]) {
    int round = 0, cthreads = 20, gthreads = 64, groupSize = 0;
    int *bufferIndex_h, *bufferIndex_d;
    value_t *result_h, *result_d;
    range_t range_h = {0, 1024}, *range_d;
    struct timeval t1, t2;
    double timeuse;
    try {
        //parse the command args
        TCLAP::CmdLine cmd("Command Description Message", ' ', "1.0");
        TCLAP::ValueArg<int> roundArg("r", "round", "Rounds of the test", false, 16, "int");
        cmd.add(roundArg);

        TCLAP::ValueArg<int> cthreadArg("t", "cpu_threads", "The number of threads of CPU utilized for testing", false, 20, "int");
        cmd.add(cthreadArg);

        TCLAP::ValueArg<int> groupSizeArg("s", "size", "The size of searching group and equals to the scale of the result array", false, 1024, "int");
        cmd.add(groupSizeArg);

        TCLAP::ValueArg<int> gthreadArg("g", "gpu_threads", "The number of threads of GPU utilized for testing", false, 64, "int");
        cmd.add(gthreadArg);

        cmd.parse(argc, argv);
        round = roundArg.getValue();
        cthreads = cthreadArg.getValue();
        groupSize = groupSizeArg.getValue();
        gthreads = gthreadArg.getValue();
    } catch(TCLAP::ArgException &e) {
        std::cerr << "error: " << e.error() << " for arg " << e.argId() << std::endl;
        exit(1);
    }

    omp_set_num_threads(cthreads);
    //printf("%d %d %d\n", round, cthreads, groupSize);
    //allocate CPU result buffer
    int groupMemSize = groupSize * sizeof(int);
    bufferIndex_h = (int*)malloc(cthreads * groupMemSize);
    result_h = (uint32_t*)malloc(cthreads * groupSize * sizeof(uint32_t));
    if (!bufferIndex_h || !result_h) {
        std::cerr << "Memory Allocation For Host Buffer has been wrong" << std::endl;
    }

    //allocate GPU result buffer
    checkCudaErrors(hipMalloc((void**)&bufferIndex_d, cthreads * groupMemSize));
    checkCudaErrors(hipMalloc((void**)&result_d, cthreads * groupSize * sizeof(uint32_t)));
    checkCudaErrors(hipMalloc((void**)&range_d, sizeof(range_t)));
    checkCudaErrors(hipMemcpy(range_d, &range_h, sizeof(range_t), hipMemcpyHostToDevice));

    
    //init the skiplist and bucket
    skiplist* sl = slCreate();
    FILE* file = fopen("keys.txt", "r");
    uint32_t key, value;
    while (!feof(file)) {
        fscanf(file, "%u %u", &key, &value);
        slInsert(sl, key, value);
    }
    BufferMemCpy(sl);

    hipStream_t* streams = (hipStream_t*)malloc(cthreads * sizeof(hipStream_t));
    if(!streams) {
        std::cerr << "Memory Allocation for streams goes wrong" << std::endl;
        exit(1);
    }
    for (int i = 0; i < cthreads; i++)
        checkCudaErrors(hipStreamCreate(&(streams[i])));

    for (int size = 1024; size <= groupSize; size *= 2) {
        gettimeofday(&t1, NULL);
        for (int r = 0; r < round; r++) {
            //global rounds of the test
            #pragma omp parallel for
            for (int t = 0; t < cthreads; t++) {
                //thread work
                getIndexArray(sl, range_h.min, range_h.max, bufferIndex_h + t * size, size);
            }
            for (int t = 0; t < cthreads; t++) {
                checkCudaErrors(hipMemcpyAsync(bufferIndex_d + t * size, bufferIndex_h + t * size, size * sizeof(int), hipMemcpyHostToDevice, streams[t]));
                bucket_search<<<size / gthreads, gthreads, 0, streams[t]>>>(range_d, result_d + t * size, sl->bucketBuffer, bufferIndex_d + t * size);
                checkCudaErrors(hipMemcpyAsync(result_h + t * size, result_d + t * size, size * sizeof(uint32_t), hipMemcpyDeviceToHost, streams[t]));
            }
        }
        checkCudaErrors(hipDeviceSynchronize());
        gettimeofday(&t2, NULL);
        timeuse = ((t2.tv_sec - t1.tv_sec)  * 1000.0 + (t2.tv_usec - t1.tv_usec)/1000.0) / round;
        std::cout << "Group Size:\t" << size << "\tTime:\t" << timeuse << std::endl;
        std::cout.flush();
    }
    slFree(sl);

    for (int i = 0; i < cthreads; i++)
        checkCudaErrors(hipStreamDestroy(streams[i]));
}
